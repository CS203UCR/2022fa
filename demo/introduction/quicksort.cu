/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc -arch=sm_11 bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS_PER_BLOCK 2048 // 2^10
#define THREADS 1024 // 2^10
//#define BLOCKS 32768 // 2^15
#define BLOCKS 32 // 2^16
#define NUM_VALS THREADS*BLOCKS
__device__ int d_size;

__global__ void partition (float *arr, int *arr_l, int *arr_h, int n)
{
    int z = blockIdx.x*blockDim.x+threadIdx.x;
    d_size = 0;
    __syncthreads();
    if (z<n)
      {
        int h = arr_h[z];
        int l = arr_l[z];
        int x = arr[h];
        int i = (l - 1);
        int temp;
        for (int j = l; j <= h- 1; j++)
          {
            if (arr[j] <= x)
              {
                i++;
                temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
              }
          }
        temp = arr[i+1];
        arr[i+1] = arr[h];
        arr[h] = temp;
        int p = (i + 1);
        if (p-1 > l)
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = l;
            arr_h[ind] = p-1;  
          }
        if ( p+1 < h )
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = p+1;
            arr_h[ind] = h; 
          }
      }
}
 
void quickSortIterative (float arr[], int l, int h)
{
    int lstack[ h - l + 1 ], hstack[ h - l + 1];
    float *d_d;
    int top = -1, *d_l, *d_h;
 
    lstack[ ++top ] = l;
    hstack[ top ] = h;

    hipMalloc(&d_d, (h-l+1)*sizeof(float));
    hipMemcpy(d_d, arr,(h-l+1)*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_l, (h-l+1)*sizeof(int));
    hipMemcpy(d_l, lstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&d_h, (h-l+1)*sizeof(int));
    hipMemcpy(d_h, hstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);
    int n_t = 1;
    int n_b = 1;
    int n_i = 1; 
    while ( n_i > 0 )
    {
        partition<<<n_b,n_t>>>( d_d, d_l, d_h, n_i);
        int answer;
        hipMemcpyFromSymbol(&answer, HIP_SYMBOL(d_size), sizeof(int), 0, hipMemcpyDeviceToHost); 
        if (answer < 1024)
          {
            n_t = answer;
          }
        else
          {
            n_t = 1024;
            n_b = answer/n_t + (answer%n_t==0?0:1);
          }
        n_i = answer;
        hipMemcpy(arr, d_d,(h-l+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
}
 

/* quicksort
 * 
 * This function is called by main to populate a result, testing the CPU
 * and GPU implementations of quicksort.
 *
 * Parameters:
 * unsorted: an unsorted array of floating points
 * length: the length of the unsorted array
 * result: an output parameter to be filled with the results of the cpu and gpu
 *         implementations of quicksort.
 *
 */
void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}


int main(void)
{
  clock_t start, stop;

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);

  start = clock();
  quickSortIterative(values, 0, NUM_VALS-1);
  stop = clock();

  print_elapsed(start, stop);
}