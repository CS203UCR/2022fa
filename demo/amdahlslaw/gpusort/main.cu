#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std; 

#define SIZE	(1 << 22)
// Number of tests to average over
#define TEST	1
// The timers for the different parts of the algo
StopWatchInterface  *uploadTimer, *downloadTimer, *bucketTimer, 
			 *mergeTimer, *totalTimer, *cpuTimer; 

// Compare method for CPU sort
inline int compare(const void *a, const void *b) {
	if(*((float *)a) < *((float *)b)) return -1; 
	else if(*((float *)a) > *((float *)b)) return 1; 
	else return 0; 
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
//#define BLOCKS 32768 // 2^15
#define THREADS_PER_BLOCK 1024 //

void bitonic_sort(float *values, float *out_values, int num_of_elements)
{
  float *dev_values;
  size_t size = num_of_elements * sizeof(float);
  int BLOCKS = num_of_elements/THREADS_PER_BLOCK; 

  hipMalloc((void**) &dev_values, size);
  sdkStartTimer(&uploadTimer);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  sdkStopTimer(&uploadTimer); 
  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS_PER_BLOCK,1);  /* Number of threads  */

  sdkStartTimer(&totalTimer); 
  int j, k;
  /* Major step */
  for (k = 2; k <= num_of_elements; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
		hipDeviceSynchronize(); 
  sdkStopTimer(&totalTimer); 
  sdkStartTimer(&downloadTimer);
  hipMemcpy((void *)out_values, (void *)dev_values,  num_of_elements*sizeof(float), hipMemcpyDeviceToHost);
//		checkCudaErrors(hipMemcpy((void *)out_values, (void *)dev_values, num_of_elements*sizeof(float), hipMemcpyDeviceToHost));
  sdkStopTimer(&downloadTimer); 
  hipFree(dev_values);
}

 #define N		THREADS_PER_BLOCK
 // Kernel function
 __global__ static void quicksort(float* values) {
 #define MAX_LEVELS	300

	int pivot, L, R;
	int idx =  threadIdx.x + blockIdx.x * blockDim.x;
	int start[MAX_LEVELS];
	int end[MAX_LEVELS];

	start[idx] = idx;
	end[idx] = N - 1;
	while (idx >= 0) {
		L = start[idx];
		R = end[idx];
		if (L < R) {
			pivot = values[L];
			while (L < R) {
				while (values[R] >= pivot && L < R)
					R--;
				if(L < R)
					values[L++] = values[R];
				while (values[L] < pivot && L < R)
					L++;
				if (L < R)
					values[R--] = values[L];
			}
			values[L] = pivot;
			start[idx + 1] = L + 1;
			end[idx + 1] = end[idx];
			end[idx++] = L;
			if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1]) {
	                        // swap start[idx] and start[idx-1]
        	                int tmp = start[idx];
                	        start[idx] = start[idx - 1];
                        	start[idx - 1] = tmp;

	                        // swap end[idx] and end[idx-1]
        	                tmp = end[idx];
                	        end[idx] = end[idx - 1];
                        	end[idx - 1] = tmp;
	                }

		}
		else
			idx--;
	}
}


typedef struct vars{
  int l;
  int r;
  int leq;
} vars;

/***************************** GPU IMPLEMENTATION ****************************/

/* gpuPartitionSwap
 *
 * This kernel function is called recursively by the host. Its purpose is to, 
 * given a pivot value, partition and swap items in the section of the input
 * array bounded by the l & r indices, then store the pivot in the correct
 * location.
 *
 * Parameters:
 * input: The unsorted (or partially sorted) input data
 * output: The aptly named output parameter, it is the same as input, but all
 *         floating points within (l,r) have been partitioned and swapped.
 * endpts: This is a custom data struct meant to 
 *         a) hold a counter variable in global memory
 *         b) pass the l' and r' parameters back to the host to the left and
 *            right of the positioned pivot item.
 * pivot: This is the pivot value, about which all items in (l,r) are being
 *        swapped.
 * l: the left index bound on input & output
 * r: the right index bound on input & output
 * d_leq: an array of offset values, storedin global device memory
 * nBlocks: The total number of blocks, to be used to determine the location
 *          of insertion of the pivot.
 *
 */
__global__ void gpuPartitionSwap(float * input, float * output, vars * endpts, 
				 float pivot, int l, int r, 
                                 int d_leq[], 
				 int d_gt[], int *d_leq_val, int *d_gt_val, 
                                 int nBlocks)
{
  //copy a section of the input into shared memory
  __shared__ float bInput[THREADS_PER_BLOCK];
  __syncthreads();
  int idx = l + blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;
  __shared__ int lThisBlock, rThisBlock;
  __shared__ int lOffset, rOffset;

  if(threadIdx.x == 0){
    d_leq[blockIdx.x] = 0;
    d_gt[blockIdx.x] = 0;
    *d_leq_val = 0;
    *d_gt_val = 0;
    //cuPrintf("STARTING NEXT SET!  l: %d, r: %d\n", l, r);
  }
  __syncthreads();

  if(idx <= (r - 1)){
    bInput[threadIdx.x] = input[idx];

    //make comparison against the pivot, setting 'status' and updating the counter (if necessary)
    if( bInput[threadIdx.x] <= pivot ){
        //cuPrintf("input:  %f < pivot: %f\n", bInput[threadIdx.x], pivot);
        atomicAdd( &(d_leq[blockIdx.x]), 1);
    } else {
        //cuPrintf("input:  %f > pivot: %f\n", bInput[threadIdx.x], pivot);
        atomicAdd( &(d_gt[blockIdx.x]), 1);
    }
    
  }
  __syncthreads();


  if (threadIdx.x == 0){
      lThisBlock = d_leq[blockIdx.x];
      lOffset = l+atomicAdd(d_leq_val, lThisBlock);
  }
  if (threadIdx.x == 1){
      rThisBlock = d_gt[blockIdx.x];
      rOffset = r-atomicAdd(d_gt_val, rThisBlock);
  }

  __syncthreads();

  if(threadIdx.x == 0){
    int m = 0;
    int n = 0;
    for(int j = 0; j < THREADS_PER_BLOCK; j++){
      int chk = l + blockIdx.x*THREADS_PER_BLOCK + j;
      if(chk <= (r-1) ){
	if(bInput[j] <= pivot){
	  output[lOffset + m] = bInput[j];
	  ++m;
	} else {
	  output[rOffset - n] = bInput[j];
	  ++n;
	}
      }
    }
  }

  __syncthreads();

  if((blockIdx.x == 0) && (threadIdx.x == 0)){
    int pOffset = l;
    for(int k = 0; k < nBlocks; k++)
      pOffset += d_leq[k];

    output[pOffset] = pivot;
    endpts->l = (pOffset - 1);
    endpts->r = (pOffset + 1);
  }

  return;
}

void gqSort(float ls[], int l, int r, int length){
  //if (r - l) >= 1
  if((r - l) >= 1){


    //1. grab pivot
    float pivot = ls[r];

    //2. set-up gpu vars
    int numBlocks = (r - l) / THREADS_PER_BLOCK;
    if((numBlocks * THREADS_PER_BLOCK) < (r - l))
      numBlocks++;

    float * d_ls;
    float * d_ls2;
    vars endpts;
    endpts.l = l;
    endpts.r = r;

    vars * d_endpts;
    int * d_leq, * d_gt, *d_leq_val, *d_gt_val;
    int size = sizeof(float);
    hipMalloc(&(d_ls), size*length);
    hipMalloc(&(d_ls2), size*length);
    hipMalloc(&(d_endpts), sizeof(vars));
    hipMalloc(&(d_leq), 4*numBlocks);
    hipMalloc(&(d_gt), 4*numBlocks);
    hipMalloc(&d_leq_val, 4);
    hipMalloc(&d_gt_val, 4);
//    sdkStartTimer(&uploadTimer); 

    hipMemcpy(d_ls, ls, size*length, hipMemcpyHostToDevice);
    hipMemcpy(d_ls2, ls, size*length, hipMemcpyHostToDevice);
//    sdkStopTimer(&downloadTimer); 

    //3. call gpuPartition function
//    sdkStartTimer(&totalTimer); 
    gpuPartitionSwap<<<numBlocks, THREADS_PER_BLOCK>>>(d_ls, d_ls2, d_endpts, pivot, l, r, d_leq, d_gt, d_leq_val, d_gt_val, numBlocks);
//    sdkStopTimer(&totalTimer); 

    //4. Retrieve sorted list and other variables
    hipMemcpy(ls, d_ls2, size*length, hipMemcpyDeviceToHost);
    hipMemcpy(&(endpts), d_endpts, sizeof(vars), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    //5.recursively call on left/right sections of list generated by gpuPartition

    hipFree(d_ls);
    hipFree(d_ls2);
    hipFree(d_endpts);
    hipFree(d_leq);
    hipFree(d_gt);

    if(endpts.l >= l)
      gqSort(ls, l, endpts.l, length);
    if(endpts.r <= r)
      gqSort(ls, endpts.r, r, length);
  
  }

  return;
}

/* gpu_quicksort
 *
 * This is a function meant to set up the custom 'data' struct array
 * used by the gpu implementation of quicksort, as well as to calculate
 * the time of execution of the sorting algorithm.
 *
 * Parameters:
 * unsorted: The array of floats to be sorted
 * length: The length of the unsorted and sorted arrays
 * sorted: An output parameter, to be filled with the sorted array.
 *
 * Output:
 * time: This function returns the time of execution required by the
 *       sorting algorithm
 */
double gpu_quicksort(float unsorted[], float sorted[], int length){
  time_t start, end;
  double time;
  double *dev_values;
//  for(int i = 0; i < length; i++)
//    sorted[i] = unsorted[i];

  hipMalloc((void**) &dev_values, length);
  sdkStartTimer(&uploadTimer);
  hipMemcpy(dev_values, unsorted, length, hipMemcpyHostToDevice);
  sdkStopTimer(&uploadTimer); 
  sdkStartTimer(&totalTimer); 
  quicksort <<< length / THREADS_PER_BLOCK, length / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (unsorted);
//  gqSort(sorted, 0, length - 1, length);
  end = clock();
  time = ((double) end - start) / CLOCKS_PER_SEC;
  hipDeviceSynchronize(); 
  sdkStopTimer(&totalTimer); 
  sdkStartTimer(&downloadTimer);
  hipMemcpy((void *)sorted, (void *)dev_values, length*sizeof(float), hipMemcpyDeviceToHost);
  sdkStopTimer(&downloadTimer); 
  hipFree(dev_values);
  return time;
}
#define SHAREDSIZE 8000  /* Should be changed to dynamically detect shared
                             memory size if at all possible.  */

//Forward declarations
__global__ void GPUMerge(float *d_list, int len, int stepSize,
                         int eltsPerThread);

/* Mergesort definition.  Takes a pointer to a list of floats.
 * the length of the list, the number of threads per block, and 
 * the number of blocks on which to execute.  
 * Puts the list into sorted order in-place.*/
void MergeSort(float *h_list, int len, int threadsPerBlock, int blocks) {

    float *d_list;
    if ( (hipMalloc((void **) &d_list, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return;
    }

  sdkStartTimer(&uploadTimer);
    hipMemcpy(d_list, h_list, len*sizeof(float), hipMemcpyHostToDevice);
  sdkStopTimer(&uploadTimer);

    int stepSize = ceil(len/float(threadsPerBlock*blocks));
    int eltsPerThread = ceil(stepSize/threadsPerBlock);
    int maxStep = SHAREDSIZE/sizeof(float);

    if (maxStep < stepSize) {
        stepSize = maxStep;
    }

  sdkStartTimer(&totalTimer);
    GPUMerge<<<blocks, threadsPerBlock>>>(d_list, len, stepSize,
                                          eltsPerThread);
  sdkStopTimer(&totalTimer);

  sdkStartTimer(&downloadTimer);
    hipMemcpy(h_list, d_list, len*sizeof(float), hipMemcpyDeviceToHost);
  sdkStopTimer(&downloadTimer);
    hipFree(d_list);
}

/* Mergesort definition.  Takes a pointer to a list of floats, the length
 * of the list, and the number of list elements given to each thread.
 * Puts the list into sorted order in-place.*/
__global__ void GPUMerge(float *d_list, int len, int stepSize,
                         int eltsPerThread){

    int my_start, my_end; //indices of each thread's start/end

    //Declare counters requierd for recursive mergesort
    int l_start, r_start; //Start index of the two lists being merged
    int old_l_start;
    int l_end, r_end; //End index of the two lists being merged
    int headLoc; //current location of the write head on the newList
    short curList = 0; /* Will be used to determine which of two lists is the
                        * most up-to-date, since merge sort is not an in-place
                        * sorting algorithm. */

    //Attempt to allocate enough shared memory for this block's list...
    //Note that mergesort is not an in-place sort, so we need double memory.
    __shared__ float subList[2][SHAREDSIZE/sizeof(float)];

    //Load memory
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for (int i = 0; i < eltsPerThread; i++){
        if (index + i < len){
            subList[curList][eltsPerThread*threadIdx.x + i]=d_list[index + i];
        }
    }

    //Wait until all memory has been loaded
    __syncthreads();
    
    //Merge the left and right lists.
    for (int walkLen = 1; walkLen < len; walkLen *= 2) { 
        //Set up start and end indices.
        my_start = eltsPerThread*threadIdx.x;
        my_end = my_start + eltsPerThread;
        l_start = my_start;

        while (l_start < my_end) { 
            old_l_start = l_start; //l_start will be getting incremented soon.
            //If this happens, we are done.
            if (l_start > my_end){
                l_start = len;
                break;
            }
            
            l_end = l_start + walkLen;
            if (l_end > my_end) {
                l_end = len;
            }
            
            r_start = l_end;
            if (r_start > my_end) {
                r_end = len;
            }
            
            r_end = r_start + walkLen;
            if (r_end > my_end) {
                r_end = len;
            }
            
            for (int i = 0; i < walkLen; i++){
                if (subList[curList][l_start] < subList[curList][r_start]) {
                    subList[!curList][headLoc] = subList[curList][l_start];
                    l_start++;
                    headLoc++;
                    //Check if l is now empty
                    if (l_start == l_end) {
                        for (int j = r_start; j < r_end; j++){
                            subList[!curList][headLoc] = 
                                subList[curList][r_start];
                            r_start++;
                            headLoc++;
                    }
                    } 
                }
                else {
                    subList[!curList][headLoc] = subList[curList][r_start];
                    r_start++;
                    //Check if r is now empty
                    if (r_start == r_end) {
                        for (int j = l_start; j < l_end; j++){
                            subList[!curList][headLoc] = 
                                subList[curList][r_start];
                            r_start++;
                            headLoc++;
                        }
                    } 
                }
            }

            l_start = old_l_start + 2*walkLen;
            curList = !curList;
        }
    }
    
    return;

    //subList[blockIdx

    //...otherwise, we use global memory...
    /*
    if ( (subList = hipMalloc(stepsize*sizeof(float)) != NULL ) {
            //   do some shit.
            
        }    
    */

    //...otherwise, we give up.

}


int main( int argc, char** argv)
{ 

  // Create timers for each sort
    struct timeval time_start, time_end, program_start, program_end;
    double input_time=0;
    gettimeofday(&program_start, NULL);	

    sdkCreateTimer(&uploadTimer);
    sdkCreateTimer(&downloadTimer);
    sdkCreateTimer(&bucketTimer);
    sdkCreateTimer(&mergeTimer);
    sdkCreateTimer(&totalTimer);
    sdkCreateTimer(&cpuTimer);
	int numElements = 0;
	int method=0;
    // Number of elements in the test bed
    	gettimeofday(&time_start, NULL);
#ifdef READING_FROM_BINARY
        numElements = atoi(argv[2]);
	if (argc == 4)
		method = atoi(argv[3]);
#else
       	if(strcmp(argv[1],"r") ==0) {
	         numElements = SIZE; 
	    }
	else {
		FILE *fp;
	    fp = fopen(argv[1],"r");
	    if(fp == NULL) {
	        cout << "Error reading file" << endl;
	        exit(EXIT_FAILURE);
	    }
	    int count = 0;
	    float c;

	    while(fscanf(fp,"%f",&c) != EOF) {
	        count++;
        }
	    fclose(fp);
	    numElements = count;
    }
#endif

	cout << "Sorting list of " << numElements << " floats\n";
	// Generate random data
	// Memory space the list of random floats will take up
	int mem_size = numElements * sizeof(float); 
	// Allocate enough for the input list
	float *cpu_idata = (float *)malloc(mem_size);
	// Allocate enough for the output list on the cpu side
	float *cpu_odata = (float *)malloc(mem_size);
	// Allocate enough memory for the output list on the gpu side
	float *gpu_odata = (float *)malloc(mem_size);

	float datamin = FLT_MAX; 
	float datamax = -FLT_MAX; 
	if(strcmp(argv[1],"r")==0) {

	for (int i = 0; i < numElements; i++) {
	// Generate random floats between 0 and 1 for the input data
		cpu_idata[i] = ((float) rand() / RAND_MAX); 
	//Compare data at index to data minimum, if less than current minimum, set that element as new minimum
		datamin = min(cpu_idata[i], datamin);
	//Same as above but for maximum
		datamax = max(cpu_idata[i], datamax);
	}
    }	else {
	FILE *fp;
#ifdef READING_FROM_BINARY
	fp = fopen(argv[1],"rb");
	fread(cpu_idata,sizeof(float),numElements,fp);
	fclose(fp);
   	gettimeofday(&time_end, NULL);
    input_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0;
	printf("FileInput %lf seconds\n",input_time);
	
    /*	for(int i = 0; i < numElements; i++) {
	datamin = min(cpu_idata[i],datamin);
	datamax = max(cpu_idata[i],datamax);
	}*/

	datamin = 0.0;
	datamax = 16777215.0;
#else
	fp = fopen(argv[1],"r");
	for(int i = 0; i < numElements; i++) {
	fscanf(fp,"%f",&cpu_idata[i]);
	datamin = min(cpu_idata[i],datamin);
	datamax = max(cpu_idata[i],datamax);
	}
	fclose(fp);
   	gettimeofday(&time_end, NULL);
    input_time = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0;
	printf("FileInput %lf seconds\n",input_time);
    	gettimeofday(&time_start, NULL);	
#endif
	}
#ifndef CPU
	cout << "Sorting on GPU..." << flush; 
	// GPU Sort
	if(method == 0)
	for (int i = 0; i < TEST; i++) 
        MergeSort(cpu_idata, numElements, THREADS_PER_BLOCK, numElements/THREADS_PER_BLOCK);
//		gpu_quicksort(cpu_idata, gpu_odata, numElements);		
	else {
	// Using bitonic sort
	for (int i = 0; i < TEST; i++) 
		bitonic_sort(cpu_idata, gpu_odata, numElements);		
	}
#endif

#if defined(VERIFY) || (defined CPU)
	cout << "Sorting on CPU..." << flush; 
	// Start_CPU_Sort
	sdkStartTimer(&cpuTimer); 
	memcpy(cpu_odata, cpu_idata, mem_size);
    qsort(cpu_odata, numElements, sizeof(float), compare);
	sdkStopTimer(&cpuTimer); 
    // End_CPU_Sort 
	cout << "done.\n";
	#ifdef VERIFY
    cout << "Checking result..." << flush; 
	// Result checking
	int count = 0; 
	for(int i = 0; i < numElements; i++)
		if(fabs(cpu_odata[i] - gpu_odata[i])> 0.0002)
		{
			printf("Sort missmatch on element %d: \n", i); 
			printf("CPU = %f : GPU = %f\n", cpu_odata[i], gpu_odata[i]); 
			count++; 
			break; 
		}
	if(count == 0) cout << "PASSED.\n";
	else cout << "FAILED.\n";
    #endif
#endif
#ifndef CPU
// Timer report
	printf("GPU iterations: %d\n", TEST); 
#endif
#ifdef TIMER
#if defined(VERIFY) || (defined CPU)
	printf("Total CPU execution time: %lf seconds\n", sdkGetTimerValue(&cpuTimer)/1000);
#endif
#ifndef CPU
	printf("Total GPU Sort execution time: %lf seconds\n", sdkGetTimerValue(&totalTimer) /1000);
	printf("    - Upload		: %lf seconds\n", sdkGetTimerValue(&uploadTimer) /1000);
	printf("    - Download		: %lf seconds\n", sdkGetTimerValue(&downloadTimer) /1000);
//	printf("    - Bucket sort	: %lf seconds\n", sdkGetTimerValue(&bucketTimer) /1000);
//	printf("    - Merge sort	: %lf seconds\n", sdkGetTimerValue(&mergeTimer) /1000);
#endif
#endif

	
    // Release memory
    gettimeofday(&program_end, NULL);
    double total_time = ((program_end.tv_sec * 1000000 + program_end.tv_usec) - (program_start.tv_sec * 1000000 + program_start.tv_usec))/1000000.0;
    fprintf(stderr, "%lf, %lf, %lf, %lf, %lf, %lf\n", total_time, input_time, sdkGetTimerValue(&cpuTimer)/1000, sdkGetTimerValue(&totalTimer) /1000, sdkGetTimerValue(&uploadTimer) /1000, sdkGetTimerValue(&downloadTimer) /1000);
    sdkDeleteTimer(&uploadTimer);
    sdkDeleteTimer(&downloadTimer);
    sdkDeleteTimer(&bucketTimer);
    sdkDeleteTimer(&mergeTimer);
    sdkDeleteTimer(&totalTimer);
    sdkDeleteTimer(&cpuTimer);
	free(cpu_idata); 
}
