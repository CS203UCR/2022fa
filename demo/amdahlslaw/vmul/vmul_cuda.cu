#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>
#include <inttypes.h>
#include <sys/time.h>
extern "C" {
#include "vmul.h"
}

__global__ void vector_mul_double_cuda_kernel(double *va, double *vb, int size, int grid_size, int iter);

extern void vector_mul_double_cuda(double *va, double *vb, unsigned long int size, int p, int iter)
{
    double *d_va, *d_vb;
    struct timeval time_start, time_end, program_start, program_end;
    gettimeofday(&time_start, NULL);
    hipMalloc((void **) &d_va, sizeof(double)*size);
    hipMalloc((void **) &d_vb, sizeof(double)*size);
    hipMemcpy(d_va, va, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_vb, vb, sizeof(double)*size, hipMemcpyHostToDevice);
    gettimeofday(&time_end, NULL);
    perf_stats[HOST2GPU]=((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0;
    printf("memcpy (to device): %lf seconds\n",perf_stats[HOST2GPU]);
    gettimeofday(&time_start, NULL);

    unsigned int grid_size = 1024;
    unsigned int grid_cols = (size + grid_size - 1) / (p*1024);
    perf_stats[NUM_OF_THREADS] = grid_cols;
    dim3 dimGrid(grid_cols, 1);
    dim3 dimBlock(grid_size, 1);
	printf("Using GPU\n");
         // Launch kernel 
    vector_mul_double_cuda_kernel<<<(size + grid_size - 1) / (p*1024), 1024>>>(d_va, d_vb, size, p, iter);
    hipDeviceSynchronize();
    gettimeofday(&time_end, NULL);
    perf_stats[GPU_KERNEL] = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0;
    printf("kernel: %lf seconds\n",perf_stats[GPU_KERNEL]);
    gettimeofday(&time_start, NULL);
    hipMemcpy(va, d_va, sizeof(double)*size,
    hipMemcpyDeviceToHost);
    gettimeofday(&time_end, NULL);
    perf_stats[GPU2HOST] = ((time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec))/1000000.0;
    printf("memcpy (to host): %lf seconds\n",perf_stats[GPU2HOST]);
    hipFree(d_va);
    hipFree(d_vb);
	return;
}

__global__ void vector_mul_double_cuda_kernel(double *va, double *vb, int size, int grid_size, int iter) 
{
    unsigned int start_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j;
    for(i = start_pos*grid_size; i < (start_pos+1)*grid_size && i < size; i++)
        for (j = 0 ; j < iter; j++)
            va[i] = va[i]*vb[i];
}
